#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "common_kernels.h"

// Utilities and system includes
// #include <helper_functions.h>
// #include <hip/hip_runtime_api.h>

__global__ void conv(float* A,float *B, float *H, int hlen, int ylen,int width, int height) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // int row = blockIdx.y * blockDim.y + threadIdx.y;
  int i = col;
  if (col < ylen) {
    float sum = 0.0f;
    for (int j = 0; j < hlen; j++) {
      if (i-j*width < 0)break;
      sum = sum + (float) H[j] * A[i - j * width];
    }
    B[col] += sum;
  }
}

extern "C" void conv_wrap(float* A,float *B, float *H, int hlen, int ylen,int width, int height) {
  conv<<<(w+w%32),min(1024,(h + h%32)),0,stream>>>(A_gpu,B_gpu,H_gpu, flen, ylen,w,h);
}
// int main(int argc, char** argv) {
//   StopWatchInterface *hTimer = NULL;

//   // Use command-line specified CUDA device, otherwise use device with highest
//   // Gflops/s
//   findCudaDevice(argc, (const char **)argv);

//   sdkCreateTimer(&hTimer);

//   float *A, *B, *H;
//   float *A_gpu, *B_gpu,*H_gpu;
//   float *B_cpu;

//   int w = 10000;
//   int h = 10000; 
//   int ylen = w * h;
//   int flen = 30;
  
//   A = (float *)malloc(w * h * sizeof(float));
//   B = (float *)malloc(w * h * sizeof(float));
//   B_cpu = (float *)malloc(w * h * sizeof(float));
//   H = (float *)malloc(flen * sizeof(float));
  
//   for (int i = 0; i < flen; i++)
//     H[i] = 1.0f;
  
//   for (int i = 0; i < w; i++) {
//     for (int j = 0; j < h; j++) {
//       A[i * h +j] = (float) i * h + j;
//       B[i * h + j] = 0.0f;
//       B_cpu[i * h + j] = 0.0f;
//     }
//   }

//   hipMalloc((void **) &A_gpu, w * h * sizeof(float));
//   hipMalloc((void **) &B_gpu, w * h * sizeof(float));
//   hipMalloc((void **) &H_gpu, flen * sizeof(float));

//   hipMemcpy(A_gpu, A, w * h * sizeof(float), hipMemcpyHostToDevice);
//   hipMemcpy(B_gpu, B, w * h* sizeof(float), hipMemcpyHostToDevice);
//   hipMemcpy(H_gpu, H, flen * sizeof(float), hipMemcpyHostToDevice);
//   hipStream_t stream;

//   hipEvent_t start, stop;
//   checkCudaErrors(hipEventCreate(&start));
//   checkCudaErrors(hipEventCreate(&stop));

//   checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
//   checkCudaErrors(hipStreamSynchronize(stream));
//   checkCudaErrors(hipEventRecord(start, stream));
  
//   conv<<<(w+w%32),min(1024,(h + h%32)),0,stream>>>(A_gpu,B_gpu,H_gpu, flen, ylen,w,h);
//   // Record the stop event
//   checkCudaErrors(hipEventRecord(stop, stream));

//   // Wait for the stop event to complete
//   checkCudaErrors(hipEventSynchronize(stop));
//   float msecTotal = 0.0f;
//   checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
//   // Compute and print the performance
//   int iter = 1;
//   float msecPerMatrixMul = msecTotal/iter;
//   double flopsPerMatrixMul = 2.0 * static_cast<double>(w) *
//                              static_cast<double>(h) * static_cast<double>(flen) * iter;
//   double gigaFlops =
//       (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
//   printf(
//       "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
//       " WorkgroupSize= %u threads/block\n",
//       gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, 32);
//   checkCudaErrors(hipDeviceSynchronize());
//   sdkResetTimer(&hTimer);
//   sdkStartTimer(&hTimer);
//   cpu_convolve(A,B_cpu,H, flen, ylen,w,h);
//   sdkStopTimer(&hTimer);
//   sdkGetTimerValue(&hTimer);
//   printf("CPU Convolve: %.5fmsec\n",(float)sdkGetTimerValue(&hTimer));

//   // hipMemcpy(B, B_gpu, w * h * sizeof(float), hipMemcpyDeviceToHost);

//   // for (int i = 0; i < w; i++) {
//   //   for (int j = 0; j < h; j++) {
//   //     printf("%.2f,", A[i*h+j]);
//   //   }
//   //   printf("\n");
//   // }
//   // printf("\n");
//   // for (int i = 0; i < w; i++) {  
//   //   for (int j = 0; j < h; j++) {
//   //     printf("%.2f,", B[i*h+j]);
//   //   }
//   //   printf("\n");
//   // }
//   hipFree(A_gpu);
//   hipFree(B_gpu);
//   hipFree(H_gpu);
//   free(B_cpu);
//   free(B);
//   free(A);
//   free(H);
//   return 0;
// }