#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
// Utilities and system includes
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <fstream>
#include <string>
// #include "common_kernels.h"
// extern "C" void conv_wrap(float* A,float *B, float *H, int hlen, int ylen,int width, int height);

#define KERNEL_LENGTH 2
__constant__ float c_Kernel[KERNEL_LENGTH];

extern "C" void setConvolutionKernel(float *h_Kernel) {
  hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}
__global__ void conv(float* A,float *B, float *H, int hlen, int ylen,int width, int height, int lb, int span) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // int row = blockIdx.y * blockDim.y + threadIdx.y;
  int i = col;
  if (col < ylen && col >= lb * width) {
    float sum = 0.0f;
    for (int j = 0; j < hlen; j++) {
      if (i-j*width < 0)
        break;
      sum = sum + (float) H[j] * A[i - j * width];
    }
    B[i] = sum;
  }
}

__global__ void const_conv(float* A,float *B, int ylen,int width, int height) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // int row = blockIdx.y * blockDim.y + threadIdx.y;
  int i = col;
  if (col < ylen) {
    float sum = 0.0f;
    for (int j = 0; j < KERNEL_LENGTH; j++) {
      if (i-j*width < 0)break;
      sum = sum + (float) c_Kernel[j] * A[i - j * width];
    }
    B[col] += sum;
  }
}

extern "C" void conv_wrap(float* A,float *B, float *H, int hlen, int ylen,int width, int height,int lb,int span) {
  int threads = min(1024,(height + height%32));
  int off_t = 0;
  if (threads == 1024) {
    off_t = height - threads;
  }

  // int blocks = ceil(ylen / min(height, threads))
  conv<<<(width+off_t+width%32),min(1024,(height + height%32))>>>(A,B,H, hlen, ylen,width,height,lb,span);
}
extern "C" void const_conv_wrap(float* A,float *B, int ylen,int width, int height) {
  const_conv<<<(width+width%32),min(1024,(height + (32 - height%32)))>>>(A,B, ylen,width,height);
}


void cpu_convolve(float *A, float *B, float *H, int hlen, int ylen, int width, int height) {
  for (int r = 0; r < width; r++) {
    for (int i = 0; i < height; i++) {
      float sum = 0.0f;
      for (int j = 0; j < hlen; j++) {
        if (i - j < 0) break;
        sum = sum + H[j] * A[(i-j)*width];
      }
      B[r + i * width] += sum;
    }
  }
}

float dataloader(float* buffer) {
  // Open the binary file for reading
  std::ifstream file("frames.bin", std::ios::binary);
  if (!file.is_open()) {
      std::cerr << "Failed to open file." << std::endl;
      return 1;
  }

  // Determine the size of the file
  file.seekg(0, std::ios::end);
  std::streampos fileSize = file.tellg();
  file.seekg(0, std::ios::beg);

  // Calculate the size of each array
  std::streampos arraySize = fileSize / 600;
  buffer = (float *)malloc(fileSize * sizeof(float));
  // Read the file into 600 arrays
  std::vector<std::vector<char>> arrays(600);
  char* buf = (char*)malloc((int)arraySize*sizeof(char));
  for (int i = 0; i < 600; ++i) {
      // Resize the vector to hold the data for one array
      // std::cout << arraySize << std::endl;
      // arrays[i].resize(arraySize);
      // Read data into the vector
      file.read(buf,arraySize);
      for (int x = 0; x < arraySize; x++)
        buffer[i * arraySize+x] = (float)buf[x];
      // file.read(arrays[i].data(), arraySize);
      // Check for errors
      if (file.bad()) {
          std::cerr << "Error reading file." << std::endl;
          return 1;
      }
  }
  free(buf);

  // Close the file
  file.close();
  return fileSize;

}

int datawriter(float* buffer, float bufferSize) {
    // Open the binary file for writing
    std::ofstream file("output.bin", std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Failed to open file." << std::endl;
        return 1;
    }

    // Write the buffer to the file
    file.write(reinterpret_cast<const char*>(buffer), bufferSize * sizeof(float));
    if (!file.good()) {
        std::cerr << "Error writing to file." << std::endl;
        return 1;
    }

    // Close the file
    file.close();

    std::cout << "File written successfully." << std::endl;
    return 0;
}

int main(int argc, char** argv) {
  float* databuf = NULL;
  float bytes = dataloader(databuf);
  int count = bytes / (640*400);
  printf("%.2f",bytes);
  free(databuf);
  return 0;
  StopWatchInterface *hTimer = NULL;
  // Use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  
  findCudaDevice(argc, (const char **)argv);
  sdkCreateTimer(&hTimer);

  float *A, *B, *H;
  float *A_gpu, *B_gpu,*H_gpu;
  float *blur, *deriv, *mean;
  float *blur_gpu, *deriv_gpu, *mean_gpu;
  float *B_cpu;
  // float *databuf;
  int w = 640;
  int h = 400;
  int ylen = w * h;
  
  // float *gaussian;
  int lb = 2;
  int span = h;

  float gaussian[9] = {0.000133831,0.00443186,0.0539911,0.241971,0.398943,0.241971,0.0539911,0.00443186,0.000133831};
  int k = 9;

  blur = (float *)malloc(k * sizeof(float));
  for (int i = 0; i < k; i++)
    blur[i] = gaussian[i];
  hipMalloc((void **) &blur_gpu, k*sizeof(float));
  hipMemcpy(blur_gpu, blur,k*sizeof(float), hipMemcpyHostToDevice);

  mean = (float *)malloc(k * sizeof(float));
  for (int i = 0; i < k; i++)
    mean[i] = (float)1/k;
  hipMalloc((void **) &mean_gpu, k*sizeof(float));
  hipMemcpy(mean_gpu, mean,k*sizeof(float), hipMemcpyHostToDevice);

  deriv = (float *)malloc(2 * sizeof(float));
  deriv[0] = -1.0;
  deriv[1] = 1.0;
  hipMalloc((void **) &deriv_gpu, 2*sizeof(float));
  hipMemcpy(deriv_gpu,deriv,2*sizeof(float), hipMemcpyHostToDevice);
  // hipMemcpy(mean,mean_gpu,k*sizeof(float), hipMemcpyDeviceToHost);
  // hipMemcpy(blur,blur_gpu,k*sizeof(float), hipMemcpyDeviceToHost);
//{
  hipMalloc((void **) &A_gpu, w * h * sizeof(float));
  hipMalloc((void **) &B_gpu, w * h * sizeof(float));
  B = (float *)malloc(w * h * sizeof(float));
  hipMemcpy(B_gpu, B, w * h* sizeof(float), hipMemcpyHostToDevice);

  
  hipStream_t stream;

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  float* orig_A = A_gpu;
  float* orig_B = B_gpu;
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  checkCudaErrors(hipStreamSynchronize(stream));
  checkCudaErrors(hipEventRecord(start, stream));
  float* output_buffer = (float *)calloc(0,bytes * sizeof(float));
  for (int i = 0; i < count; i++) {
    hipMemcpy(A_gpu, &databuf[i * ylen], w * h * sizeof(float), hipMemcpyHostToDevice);
      
    // return 0;
    // setConvolutionKernel(gaussian);
    // A = (float *)malloc(w * h * sizeof(float));



    // hipMalloc((void **) &H_gpu, flen * sizeof(float));


    // hipMemcpy(H_gpu, H, flen * sizeof(float), hipMemcpyHostToDevice);
  //}

    // for (int i = 0; i < 10; i++) {
    float* stages[3] = {blur_gpu,deriv_gpu,mean_gpu};
    int flens[3] = {9,2,9};
    int windowing[3] = {0,0,0};

    float* hold = NULL;
    for (int j = 0; j < 3; j++) {
      lb = windowing[j];
      int flen = flens[j];
      float* Hgpu = stages[j];
      conv_wrap(A_gpu,B_gpu,Hgpu, flen, ylen,w,h,lb,span);
      if (j < 2)
        hold = A_gpu;
        A_gpu = B_gpu;
        B_gpu = hold;
    }

    // checkCudaErrors(hipDeviceSynchronize());
    // sdkResetTimer(&hTimer);
    // sdkStartTimer(&hTimer);
    // cpu_convolve(A,B_cpu,H, flen, ylen,w,h);
    // sdkStopTimer(&hTimer);
    // sdkGetTimerValue(&hTimer);
    // printf("CPU Convolve: %.5fmsec\n",(float)sdkGetTimerValue(&hTimer));

    hipMemcpy(&output_buffer[(i * ylen)], B_gpu, w * h * sizeof(float), hipMemcpyDeviceToHost);
  }
  datawriter(output_buffer, bytes);
  free(output_buffer);
  free(databuf);
    // const_conv_wrap(A_gpu,B_gpu,ylen,w,h);
  // Record the stop event
  checkCudaErrors(hipEventRecord(stop, stream));

//  Wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));
  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  // Compute and print the performance
  int iter = 1;
  float msecPerMatrixMul = msecTotal/iter;
  double flopsPerMatrixMul = 2.0 * static_cast<double>(w) *
                            static_cast<double>(h) * static_cast<double>(20) * iter;
  double gigaFlops =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
  printf(
      "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
      " WorkgroupSize= %u threads/block\n",
      gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, 32);
  // hipMemcpy(A, orig_A, w * h * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(deriv,deriv_gpu,2*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(mean,mean_gpu,k*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(blur,blur_gpu,k*sizeof(float), hipMemcpyDeviceToHost);

  // for (int i = 0; i < row; i++) {
  //   for (int j = 0; j < col; j++) {
  //     printf("%.2f\t", A[i*col+j]);
  //   }
  //   printf("\n");
  // }
  // printf("\n");
  // for (int i = 0; i < row; i++) {  
  //   for (int j = 0; j < col; j++) {
  //     printf("%.2f\t", B[i*col+j]);
  //   }
  //   printf("\n");
  // }
  hipFree(orig_A);
  // free(A);
  hipFree(orig_B);
  // free(B);
  // hipFree(H_gpu);
  
  hipFree(blur_gpu);
  free(blur);

  hipFree(deriv_gpu);
  free(deriv);
  
  hipFree(mean_gpu);
  free(mean);
  // free(B_cpu);
  
  // free(H);
  return 0;
}